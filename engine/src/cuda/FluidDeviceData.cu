#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/unique.h>

#include "../../include/cuda/FluidDeviceData.cuh"
#include "../../include/cuda/cuda_util.h"
#include "../../include/cuda/kernels/fluid_kernels.cuh"

using namespace PhysicsEngine;
using namespace FluidKernels;

void PhysicsEngine::allocateFluidDeviceData(FluidDeviceData* fluid)
{
	int numParticles = fluid->numParticles;
	int numCells = fluid->numCells;

	// allocate memory on host
	fluid->h_pos = new float4[numParticles];
	fluid->h_vel = new float4[numParticles];
	fluid->h_acc = new float4[numParticles];
	fluid->h_spos = new float4[numParticles];
	fluid->h_svel = new float4[numParticles];

	fluid->h_rho = new float[numParticles];
	fluid->h_rho0 = new float[numParticles];
	fluid->h_pres = new float[numParticles];
	
	fluid->h_cellStartIndex = new int[numCells];
	fluid->h_cellEndIndex = new int[numCells];
	fluid->h_cellIndex = new int[numParticles];
	fluid->h_particleIndex = new int[numParticles];
	fluid->h_particleType = new int[numParticles];
	fluid->h_sparticleType = new int[numParticles];

	// allocate memory on device
	 gpuErrchk(hipMalloc((void**)&(fluid->d_pos), numParticles*sizeof(float4)));
	 gpuErrchk(hipMalloc((void**)&(fluid->d_vel), numParticles*sizeof(float4)));
	 gpuErrchk(hipMalloc((void**)&(fluid->d_acc), numParticles*sizeof(float4)));
	 gpuErrchk(hipMalloc((void**)&(fluid->d_spos), numParticles*sizeof(float4)));
	 gpuErrchk(hipMalloc((void**)&(fluid->d_svel), numParticles*sizeof(float4)));

	 gpuErrchk(hipMalloc((void**)&(fluid->d_rho), numParticles*sizeof(float)));
	 gpuErrchk(hipMalloc((void**)&(fluid->d_rho0), numParticles*sizeof(float)));
	 gpuErrchk(hipMalloc((void**)&(fluid->d_pres), numParticles*sizeof(float)));
	 gpuErrchk(hipMalloc((void**)&(fluid->d_output), 3*numParticles*sizeof(float)));

	 gpuErrchk(hipMalloc((void**)&(fluid->d_cellStartIndex), numCells*sizeof(int)));
	 gpuErrchk(hipMalloc((void**)&(fluid->d_cellEndIndex), numCells*sizeof(int)));
	 gpuErrchk(hipMalloc((void**)&(fluid->d_cellHash), numParticles*sizeof(int)));
	 gpuErrchk(hipMalloc((void**)&(fluid->d_particleIndex), numParticles*sizeof(int)));
	 gpuErrchk(hipMalloc((void**)&(fluid->d_particleType), numParticles*sizeof(int)));
	 gpuErrchk(hipMalloc((void**)&(fluid->d_sparticleType), numParticles*sizeof(int)));
}

void PhysicsEngine::deallocateFluidDeviceData(FluidDeviceData* fluid)
{
	// free memory on host
	delete [] fluid->h_pos;
	delete [] fluid->h_vel;
	delete [] fluid->h_acc;
	delete [] fluid->h_spos;
	delete [] fluid->h_svel;

	delete [] fluid->h_rho;
	delete [] fluid->h_rho0;
	delete [] fluid->h_pres;

	delete [] fluid->h_cellStartIndex;
	delete [] fluid->h_cellEndIndex;
	delete [] fluid->h_cellIndex;
	delete [] fluid->h_particleIndex;
	delete [] fluid->h_particleType;
	delete [] fluid->h_sparticleType;

	// free memory on device
	gpuErrchk(hipFree(fluid->d_pos));
	gpuErrchk(hipFree(fluid->d_vel));
	gpuErrchk(hipFree(fluid->d_acc));
	gpuErrchk(hipFree(fluid->d_spos));
	gpuErrchk(hipFree(fluid->d_svel));

	gpuErrchk(hipFree(fluid->d_rho));
	gpuErrchk(hipFree(fluid->d_rho0));
	gpuErrchk(hipFree(fluid->d_pres));
	gpuErrchk(hipFree(fluid->d_output));

	gpuErrchk(hipFree(fluid->d_cellStartIndex));
	gpuErrchk(hipFree(fluid->d_cellEndIndex));
	gpuErrchk(hipFree(fluid->d_cellHash));
	gpuErrchk(hipFree(fluid->d_particleIndex));
	gpuErrchk(hipFree(fluid->d_particleType));
	gpuErrchk(hipFree(fluid->d_sparticleType));
}

void PhysicsEngine::initializeFluidDeviceData(FluidDeviceData* fluid)
{
	// numParticles = particles.size() / 3;

	// //numFluidParticles = particles.size() / 3;

	// dt = 0.0075f;
	// kappa = 1.0f;
	// rho0 = 1000.0f;
	// mass = 0.01f;

	// h = grid->getDx();
	// h2 = h * h;
	// h6 = h2 * h2 * h2;
	// h9 = h6 * h2 * h;

	// numCells = grid->getNx() * grid->getNy() * grid->getNz();

	// particleGridDim.x = grid->getNx();
	// particleGridDim.y = grid->getNy();
	// particleGridDim.z = grid->getNz();

	// particleGridSize.x = grid->getX();
	// particleGridSize.y = grid->getY();
	// particleGridSize.z = grid->getZ();

	// allocateMemory();

	// for (int i = 0; i < numParticles; i++){
	// 	h_pos[i].x = particles[3 * i];
	// 	h_pos[i].y = particles[3 * i + 1];
	// 	h_pos[i].z = particles[3 * i + 2];
	// 	h_pos[i].w = 0.0f;

	// 	h_vel[i].x = 0.0f;
	// 	h_vel[i].y = 0.0f;
	// 	h_vel[i].z = 0.0f;
	// 	h_vel[i].w = 0.0f;

	// 	h_rho0[i] = rho0;

	// 	h_particleType[i] = particleTypes[i];
	// }
	
	// gpuErrchk(hipMemcpy(d_pos, h_pos, numParticles*sizeof(float4), hipMemcpyHostToDevice));
	// gpuErrchk(hipMemcpy(d_vel, h_vel, numParticles*sizeof(float4), hipMemcpyHostToDevice));
	// gpuErrchk(hipMemcpy(d_rho0, h_rho0, numParticles*sizeof(float), hipMemcpyHostToDevice));
	// gpuErrchk(hipMemcpy(d_particleType, h_particleType, numParticles*sizeof(int), hipMemcpyHostToDevice));

	// initCalled = true;
}

void PhysicsEngine::updateFluidDeviceData(FluidDeviceData* fluid)
{
	dim3 gridSize(256,1,1);
	dim3 blockSize(256,1,1);
	//set_array_to_value<int> <<< gridSize, blockSize >>>(fluid->d_cellStartIndex, -1, fluid->numCells);
	//set_array_to_value<int> <<< gridSize, blockSize >>>(fluid->d_cellEndIndex, -1, fluid->numCells);

	build_spatial_grid <<< gridSize, blockSize >>>
	(
		fluid->d_pos, 
		fluid->d_particleIndex, 
		fluid->d_cellHash, 
		fluid->numParticles, 
		fluid->particleGridDim,
		fluid->particleGridSize
	);

	thrust::device_ptr<int> t_a(fluid->d_cellHash);
	thrust::device_ptr<int> t_b(fluid->d_particleIndex);
	thrust::sort_by_key(t_a, t_a + fluid->numParticles, t_b);

	reorder_particles <<< gridSize, blockSize >>>
	(
		fluid->d_pos,
		fluid->d_spos,
		fluid->d_vel,
		fluid->d_svel,
		fluid->d_particleType,
		fluid->d_sparticleType,
		fluid->d_cellStartIndex,
		fluid->d_cellEndIndex,
		fluid->d_cellHash,
		fluid->d_particleIndex,
		fluid->numParticles
	);

	calculate_fluid_particle_density <<< gridSize, blockSize >>>
	(
		fluid->d_spos,  
		fluid->d_rho, 
		fluid->d_sparticleType,
		fluid->d_cellStartIndex,
		fluid->d_cellEndIndex,
		fluid->d_cellHash,
		fluid->d_particleIndex,
		fluid->numParticles,
		fluid->h2,
		fluid->h9,
		fluid->particleGridDim
	);

	calculate_solid_particle_density <<< gridSize, blockSize >>>
	(
		fluid->d_spos,
		fluid->d_rho,
		fluid->d_sparticleType,
		fluid->d_cellStartIndex,
		fluid->d_cellEndIndex,
		fluid->d_cellHash,
		fluid->d_particleIndex,
		fluid->numParticles,
		fluid->h2,
		fluid->h9,
		fluid->particleGridDim
	);

	calculate_pressure <<< gridSize, blockSize >>>
	(
		fluid->d_rho,
		fluid->d_rho0,
		fluid->d_pres,
		fluid->numParticles,
		fluid->kappa
	);

	apply_pressure_and_gravity_acceleration <<< gridSize, blockSize >>>
	(
		fluid->d_spos, 
		fluid->d_svel,
		fluid->d_rho,
		fluid->d_pres,
		fluid->d_sparticleType,
		fluid->d_cellStartIndex,
		fluid->d_cellEndIndex,
		fluid->d_cellHash,
		fluid->d_particleIndex,
		fluid->numParticles,
		fluid->dt,
		fluid->h,
		fluid->h6,
		fluid->particleGridDim
	);

	compute_solid_particle_velocity <<< gridSize, blockSize >>>
	(
		fluid->d_spos,
		fluid->d_svel,
		fluid->d_sparticleType,
		fluid->numParticles
	);

	apply_xsph_viscosity <<< gridSize, blockSize >>>
	(
		fluid->d_spos,
		fluid->d_svel,
		fluid->d_rho,
		fluid->d_sparticleType,
		fluid->d_cellStartIndex,
		fluid->d_cellEndIndex,
		fluid->d_cellHash,
		fluid->d_particleIndex,
		fluid->numParticles,
		fluid->dt,
		fluid->h,
		fluid->h6,
		fluid->particleGridDim
	);

	update_particles<<< gridSize, blockSize >>>
	(
		fluid->d_spos,
		fluid->d_svel,
		fluid->d_sparticleType,
		fluid->dt,
		fluid->h,
		fluid->numParticles,
		fluid->particleGridSize
	);

	copy_sph_arrays<<< gridSize, blockSize >>>
	(
		fluid->d_pos,
		fluid->d_spos,
		fluid->d_vel,
		fluid->d_svel,
		fluid->d_particleType,
		fluid->d_sparticleType,
		fluid->d_output,
		fluid->numParticles
	);

	gpuErrchk(hipMemcpy(&((fluid->particles)[0]), fluid->d_output, 3*fluid->numParticles*sizeof(float), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(fluid->h_pos, fluid->d_pos, fluid->numParticles*sizeof(float4), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(fluid->h_rho, fluid->d_rho, fluid->numParticles*sizeof(float), hipMemcpyDeviceToHost));
}